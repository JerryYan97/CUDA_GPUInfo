#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

int main()
{
	int dev_count;
	hipGetDeviceCount(&dev_count);
	std::cout << "cuda device number: " << dev_count << std::endl;
	hipDeviceProp_t dev_prop;
	for (int i = 0; i < dev_count; i++)
	{
		hipGetDeviceProperties(&dev_prop, i);
		std::cout << "max threads per block: " << dev_prop.maxThreadsPerBlock << std::endl;
		std::cout << "multi-processor count: " << dev_prop.multiProcessorCount << std::endl;
		std::cout << "clock rate: " << dev_prop.clockRate << std::endl;
	}
    return 0;
}

